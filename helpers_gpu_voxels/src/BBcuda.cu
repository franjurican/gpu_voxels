#include "hip/hip_runtime.h"
// C++ stl
#include <iostream>

// CUDA thrust
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

// Gpu-Voxels
#include <gpu_voxels/helpers/common_defines.h>

// ROS
#include <ros/ros.h>

// header
#include <helpers_gpu_voxels/bb_cuda.h>

using namespace gpu_voxels;

namespace bb_cuda
{
    // CUDA kerenel for colliding
    __global__
    void collideMapsKernel(int8_t *probMap, uint64_t *bitMap, uint32_t voxelNum, uint16_t elementsPerThread, int8_t threshold, uint16_t *out)
    {
        // linear index, count and map offset
        uint linearId = blockDim.x*blockIdx.x + threadIdx.x;
        uint16_t count = 0;
        uint32_t offset = linearId*elementsPerThread, k;

        // detect collision
        for(int i = 0; i < elementsPerThread; i++)
        {
            if((i + offset) < voxelNum)
            {
                if(probMap[i + offset] > threshold)
                {
                    k = 4*(i + offset);

                    if((bitMap[k] != 0) || (bitMap[k + 1] != 0) || (bitMap[k + 2] != 0) || (bitMap[k + 3] != 0))
                        count++;

                }
            }
            else
            {
                break;
            } 
        }

        out[linearId] = count;
    }
}

bb_cuda::VoxelCloudPtr bb_cuda::getObjectVoxelCloud(bb_cuda::ProbVoxelMapPtr map, Eigen::Vector3f leftDown, 
                            Eigen::Vector3f rightUp, Eigen::Vector3f sceneOffset, Eigen::Vector3f offset, float threshold)
{
    // local vars
    uint32_t indexLinear, dimX, dimY, mapBytesOnDevice;
    int8_t *mapValuesInt;
    float voxelSize, probabValueFloat, xm, ym, zm;
    Vector3ui ld, ur;
    bb_cuda::VoxelCloudPtr voxelCloud(new std::vector<Eigen::Vector3f>);

    // map size on device, voxel size and map dimension (voxels)
    mapBytesOnDevice = map->getMemoryUsage();
    voxelSize = map->getVoxelSideLength();
    dimX = map->getDimensions().x;
    dimY = map->getDimensions().y;

    // get search area in map frame
    leftDown += sceneOffset;
    rightUp += sceneOffset;

    // check area
    if(!bb_cuda::checkPoint(leftDown, map))
    {
        ROS_ERROR("Tocka \"leftDown\" je izvan dimenzija mape. Izalazim iz trazenja VoxelCloud-a objekta!");
        return voxelCloud;
    } 
    else if(!bb_cuda::checkPoint(rightUp, map))
    {
        ROS_ERROR("Tocka \"upRight\" je izvan dimenzija mape. Izalazim iz trazenja VoxelCloud-a objekta!");
        return voxelCloud;
    }

    // search area in voxels
    leftDown = leftDown/voxelSize;
    rightUp = rightUp/voxelSize;
    ld.x = (uint32_t)leftDown.x();
    ld.y = (uint32_t)leftDown.y();
    ld.z = (uint32_t)leftDown.z();
    ur.x = (uint32_t)rightUp.x();
    ur.y = (uint32_t)rightUp.y();
    ur.z = (uint32_t)rightUp.z();

    // search area info
    ROS_INFO_STREAM("Podrucje trazenja VoxelClud-a max.: " << "(x, y, z) = (" << ld.x << ", " << ld.y << ", " << ld.z << ")");
    ROS_INFO_STREAM("Podrucje trazenja VoxelClud-a min.: " << "(x, y, z) = (" << ur.x << ", " << ur.y << ", " << ur.z << ")");
    
    // get map from device to host
    mapValuesInt = new int8_t[mapBytesOnDevice];
    hipMemcpy(mapValuesInt, map->getConstVoidDeviceDataPtr(), map->getMemoryUsage(), hipMemcpyDeviceToHost);

    // find objects VoxelCloud
    for(uint32_t x = ld.x; x <= ur.x; x++)
        for(uint32_t y = ld.y; y <= ur.y; y++)
            for(uint32_t z = ld.z; z <= ur.z; z++) 
            {
                // linear index and probability as float
                indexLinear = x + dimX*y + dimX*dimY*z;
                probabValueFloat = bb_cuda::convertProbabValueToFloat(mapValuesInt[indexLinear]);

                // only occupied voxels
                if(probabValueFloat > threshold)
                {
                   /* // if voxel is on edge, then exit
                    if ((x == ld.x) || (x == ur.x) || (y == ld.y) || (y == ur.y) || (z == ld.z) || (z == ur.z))
                    {
                        ROS_ERROR("Objekt se nalazi izvan prostora trazenja. Izlazim iz trazenja VoxelCloud-a objekta!!");
                        voxelCloud->clear();
                        return voxelCloud;
                    }*/

                    xm = x*voxelSize - sceneOffset.x() - offset.x();
                    ym = y*voxelSize - sceneOffset.y() - offset.y();
                    zm = z*voxelSize - sceneOffset.z() - offset.z();
                    voxelCloud->push_back(Eigen::Vector3f(xm, ym, zm));
                }
            }

    
    if(voxelCloud->size() == 0)
        ROS_WARN("Prostor trazenja je PRAZAN!");
    else
        ROS_INFO_STREAM("Velicina VoxelCloud-a objekta: " << voxelCloud->size());

    // free heap
    delete mapValuesInt;

    return voxelCloud;
}

float bb_cuda::convertProbabValueToFloat(int8_t probab)
{
    int8_t max = gpu_voxels::MAX_PROBABILITY;
    int8_t min = gpu_voxels::MIN_PROBABILITY;

    return ((float)(probab - min))/(max - min);
}

bool bb_cuda::checkPoint(Eigen::Vector3f point, bb_cuda::ProbVoxelMapPtr map)
{
    float maxX, maxY, maxZ;

    maxX = map->getMetricDimensions().x;
    maxY = map->getMetricDimensions().y;
    maxZ = map->getMetricDimensions().z;

    if((point.x() >= 0) && (point.x() < maxX) && (point.y() >= 0) && (point.y() < maxY) && (point.z() >= 0) && (point.z() < maxZ))
        return true;
    else
        return false;   
}

uint32_t bb_cuda::collideMapsFastCPU(bb_cuda::ProbVoxelMapPtr probMap, bb_cuda::BitVectorVoxelMapPtr bitMap, float threshold)
{
    int8_t *buffProb, cutOff;
    uint32_t voxelNum, sizeBytes, collNum = 0, k;
    uint64_t *buffBit;

    // get probability (one byte) -> [-128, 127]
    cutOff = gpu_voxels::ProbabilisticVoxel::floatToProbability(threshold);

    // number of voxels in map and map size in bytes
    voxelNum = probMap->getVoxelMapSize();
    sizeBytes = probMap->getMemoryUsage();

    // prob map -> one byte per voxel, bit vector map 256 bits (32 bytes) per voxel
    buffProb = new int8_t[sizeBytes]; 
    buffBit = new uint64_t[sizeBytes*4];

    // get data
    hipMemcpy((void *)buffProb, probMap->getConstVoidDeviceDataPtr(), sizeBytes, hipMemcpyDeviceToHost);
    hipMemcpy((void *)buffBit, bitMap->getConstVoidDeviceDataPtr(), sizeBytes*32, hipMemcpyDeviceToHost);

    for(uint32_t i = 0; i < voxelNum; i++)
    {
        if(buffProb[i] > cutOff)
        {
            // for bit map take every 256 bits -> 32 bytes -> 4 uint64_t
            k = i*4;

            // collision happens if voxel in bit vector map is not free
            if((buffBit[k] != 0) || (buffBit[k + 1] != 0) || (buffBit[k + 2] != 0) || (buffBit[k + 3] != 0))
                collNum++;
        }
    }

    //free heap
    delete buffProb;
    delete buffBit;

    return collNum;
}

uint32_t bb_cuda::collideMapsFastGPU(ProbVoxelMapPtr probMap, BitVectorVoxelMapPtr bitMap, float threshold, int blocks, int threads)
{
    // ptrs to device data 
    int8_t *ptrProbMap = (int8_t *)probMap->getVoidDeviceDataPtr();
    uint16_t *outDevice;
    uint64_t *ptrBitMap = (uint64_t *)bitMap->getVoidDeviceDataPtr();

    // ptr to host data
    uint16_t *outHost;

    // threshold
    int8_t probThreshold = gpu_voxels::ProbabilisticVoxel::floatToProbability(threshold);

    // number of voxels in map
    uint32_t voxelNum = probMap->getVoxelMapSize();

    // voxels per thread
    uint16_t voxelsPerThread = voxelNum/(blocks*threads) + 1;
    
    // allocate device and host memory
    hipMalloc(&outDevice, blocks*threads*sizeof(uint16_t));
    outHost = new uint16_t[blocks*threads];

    // get collision
    bb_cuda::collideMapsKernel<<<blocks, threads>>>(ptrProbMap, ptrBitMap, voxelNum, voxelsPerThread, probThreshold, outDevice);

    // copy data to host
    hipMemcpy(outHost, outDevice, blocks*threads*sizeof(uint16_t), hipMemcpyDeviceToHost);

    // get number of voxels in collision
    uint32_t numberOfVoxelsInCollision = 0;
    for(int i = 0; i < blocks*threads; i++)
        numberOfVoxelsInCollision += outHost[i];

    // free memory
    hipFree(outDevice);
    delete outHost;

    return numberOfVoxelsInCollision;
}
